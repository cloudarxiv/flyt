#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>


int main(int argc, char* argv[]) {
    if (argc != 2) {
        fprintf(stderr, "Usage: %s <gpu-device-id>\n", argv[0]);
        exit(EXIT_FAILURE);
    }

    int device_id = atoi(argv[1]);

    hipDeviceProp_t dev_prop;
    
    hipError_t res = hipGetDeviceProperties(&dev_prop, device_id);


    if (res != hipSuccess) {
        fprintf(stderr, "Error getting device properties: %s\n", hipGetErrorString(res));
        exit(EXIT_FAILURE);
    }

    printf("%u", dev_prop.multiProcessorCount);

    return 0;

}